#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include "hip/hip_runtime.h"
//#include "hip/hip_runtime_api.h"

#define THREADS 32

struct matrix {
	int ncols;
	int nrows;
	double* mat;
};

void readMatrix(struct matrix* m, FILE* file);
void printMatrix(double *squared_matrix, int n, FILE* file);
__global__ void determinant(double *l, double *u, int n, double *det, int perm);
__device__ void forwardSubstitution(double *d_l, double *d_p, double *d_y, int column, int n);
__device__ void backwardSubstitution(double *d_u, double *d_y, double *d_a_inv, int column, int n);
void pivoting(double *a, double *p, int n, int *perm);
__global__ void decomposition(double *d_l, double *d_u, int n, int k);
__device__ double atomicMul(double* address, double val);
__global__ void fillInVectors(double *d_p, double *d_l, int n);
__global__ void inverse(double *d_l,double *d_p, double *d_u, double *d_a_inv, double *d_y, int n);
__host__ void checkCudaError(int linea);



/*
 * The file which contains a matrix has in its first row the dimensions
 * then using fscanf each element of the matrix is stored on the memory allocated dynamically
*/
void readMatrix(struct matrix* m, FILE* file) {
	int i, j;

	m->mat = (double*)malloc(m->ncols * m->nrows * sizeof(double));

	for (i = 0; i < m->nrows; i++) {
		for (j = 0; j < m->ncols; j++) {
			fscanf(file, "%lf", &m->mat[i * m->ncols + j]);
		}
	}
}

/* The opposite operation of readMatrix. Stores a matrix into a file, element by element */
void printMatrix(double *squared_matrix, int n, FILE* file) {
	int i, j;

	for (i = 0; i < n; i++) {
		for (j = 0; j < n; j++) {
			fprintf(file, "%lf ", squared_matrix[i * n + j]);
		}
		fprintf(file, "\n");
	}
}

//it is not implemented directly in cuda API, it was adapted from here: https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#atomic-functions
#if __CUDA_ARCH__ < 600
__device__ double atomicMul(double* address, double val){
    unsigned long long int* address_as_ull =
                              (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val *
                               __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}
#endif

/*
* Simple function to check when cuda errors occur in kernels or memory transfers. It is used only in debug mode
*/
__host__ void checkCudaError(int linea) {
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		printf("cuda error: %s alla linea %d\n", hipGetErrorString(err), linea);
		exit(-1);
	}
}

/* 
*  Fills the diagonal positions of p and l with ones in parallel. This could lead to an improvements with wide matrices.
*/
__global__ void fillInVectors(double *d_p, double *d_l, int n){
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	
	if(i<n){
		d_p[i * n + i]=1;
		d_l[i * n + i]=1;
	}
}

/* 
* Because LU decomposition is used, det M = det LU = det L * det U.
* L and U are triangular so the determinant is calculated as the product of the diagonal elements
*/
__global__ void determinant(double *l, double *u, int n, double *d_det, int perm) {
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	double det = 1;

	if(i==0){
		d_det[0]=1; //d_det initialization
	}
	__syncthreads();

	if(i<n){
		det *= l[i * n + i] * u[i * n + i];
	}

	atomicMul(d_det, det);
	d_det[0] = d_det[0]*pow(-1,perm);
}

/* 
* Even if det(M)!=0, pivoting is performed to be sure that L and U are correctly upper and lower triangular matrix. Performed by host in serial way to avoid memory overlapping.
*/
void pivoting(double *a, double *p, int n, int *perm) {
	int j, k;
	int isMaximum = 0;
	double *temp = (double*)malloc(n * sizeof(double));

	// k is column and j is row
	for (k = 0; k < n-1; k++) {
    		int imax = k;
        	for (j = k; j < n; j++) {
			if (a[j * n + k] > a[imax * n + k]) {  // finding the maximum index
				imax = j;
        		        isMaximum = 1;
        		}
        	}
        	if (isMaximum == 1) {
        		// swapping a[k] and a[imax]
			memcpy(temp, &a[k*n], n * sizeof(double));
			memcpy(&a[k*n], &a[imax*n], n * sizeof(double));
			memcpy(&a[imax*n], temp, n * sizeof(double));

			// swapping p[k] and p[imax]
			memcpy(temp, &p[k*n], n * sizeof(double));
			memcpy(&p[k*n], &p[imax*n], n * sizeof(double));
			memcpy(&p[imax*n], temp, n * sizeof(double));

			perm[0]++;
        		isMaximum = 0;
    		}
	}
	free(temp);
}

/* 
* Perf LU decomposition of matrix M to obtain matrices L (lower) and U (upper) used to resolve and equation system throud BW and FW to obtain the inverse. It could add overhead due to the for loop from which is called in main
*/
__global__ void decomposition(double *d_l, double *d_u, int n, int k) {
    int j;
    int i = blockIdx.x*blockDim.x+threadIdx.x;	
    
	if(i>=k+1 && i<n){
            	l[i * n + k] = u[i * n + k] / u[k * n + k];
            	for (j = k; j < n; j++) {
                	u[i * n + j] = u[i * n + j] - l[i * n + k] * u[k * n + j];
            	}
        }
}

/*
*  Compute the inverse similarly to the for loop in the serial main. Each thread computer a single column of the inverse matrix (as performed in the serial code). In order to avoid memory allocation and access problems it uses a matrix to store temporary results.
*/
__global__ void inverse(double *d_l,double *d_p, double *d_u, double *d_a_inv, double *d_y, int n){
	int i = blockIdx.x*blockDim.x+threadIdx.x;	
	
	if(i<n){
		forwardSubstitution(d_l, d_p, d_y, i, n);
		backwardSubstitution(d_u, d_y, d_a_inv, i, n);
	}
}

/* 
* Since L is a lower triangular matrix forward substitution is used to perform the calculus of Lx=y 
*/
__device__ void forwardSubstitution(double *d_l, double *d_p, double *d_y, int column, int n) {
	int i, j;
	double sum = 0;

	for (i = 0; i < n; i++) {
	        for (j = 0; j < i; j++) {
	            sum = sum + d_l[i * n + j] * d_y[j*n+column];
	        }
	        d_y[i*n+column] = (d_p[i * n + column] - sum) / d_l[i * n + i];
	        sum = 0;
	}
}

/* 
* Since U is an upper triangular matrix backward substitution is used to perform the calculus of Ux=y 
*/
__device__ void backwardSubstitution(double *d_u, double *d_y, double *d_a_inv, int column, int n) {
	int i, j;
	double sum;

	d_a_inv[(n-1)*n+column] = d_y[(n-1)*n+column] / d_u[(n-1) * n + (n-1)];
	
	for (i = n - 2; i >= 0; i--) {
		sum = d_y[i*n+column];
	        for (j = n - 1; j > i; j--) {
	           sum = sum - d_u[i * n + j] * d_a_inv[j*n+column];
	        }
	        d_a_inv[i*n+column] = sum / d_u[i * n + i];
	        sum = 0;
	}
}

int main(int argc, char* argv[]) {
	if(argc != 2) { //Checking parameters: 1.mat_inv.exe 2.matrix
		printf("Parameters error.\n");
		exit(1);
	}

	//printf("This program compute the inverse of a squared matrix using only one thread\nPlease wait until computation are done...\n");

	FILE *mat, *resultFile;
	clock_t t;
	struct matrix m;
	int i, perm=0;

	mat = fopen(argv[1], "r");
	fscanf(mat, "%d %d", &m.nrows, &m.ncols);
	readMatrix(&m, mat);

	if (m.nrows != m.ncols) {
		printf("ERROR: It is not possible to compute the inversion: the matrix is not squared\n");
		fclose(mat);
		free(m.mat);
		exit(1);
	}

	int n = m.nrows; //matrix order (m is squared)

	//Create pivoting and inverse matrices
	double *a_inv = (double*)malloc(n * n * sizeof(double));
	double *p = (double*)malloc(n * n * sizeof(double));
	double *l = (double*)malloc(n * n * sizeof(double));
	double *a_p = (double*)malloc(n * n * sizeof(double));
	double *u = (double*)malloc(n * n * sizeof(double));

	//Matrices initialization
	memset(a_inv, 0, n * n * sizeof(double));
	memcpy(a_p, m.mat, n * n * sizeof(double));


	/* Device variable, allocations, and transfers */
	double *d_l, *d_u, *d_p, *d_det, *d_a_inv, *d_y;
	double det;

	hipMalloc((void**)&d_a_inv, n*n*sizeof(double));
	hipMalloc((void**)&d_p, n*n*sizeof(double));
	hipMalloc((void**)&d_l, n*n*sizeof(double));
	hipMalloc((void**)&d_u, n*n*sizeof(double));
   	hipMalloc((void**)&d_det, sizeof(double));
   	hipMalloc((void**)&d_y, n*n*sizeof(double));

	hipMemset(d_p, 0, n * n * sizeof(double));
	hipMemset(d_l, 0, n * n * sizeof(double));

	//the number of rows or columns is divided among blocks in order to give an iteration of the serial for loops to a single thread. It simply use one direction (x). Each block has THREADS number of threads
	int block_x = n / THREADS;
	if ((n) % THREADS != 0) {
		block_x++;
	}

	dim3 dimBlockLinear(THREADS, 1, 1);
	dim3 dimGridLinear(block_x, 1, 1);
	
	fillInVectors <<<dimGridLinear, dimBlockLinear>>>(d_p, d_l, n);
	
	hipMemcpy(p, d_p, n*n*sizeof(double), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	
	t = clock();
	pivoting(a_p, p, n, &perm);

	hipMemcpy(d_u, a_p, n * n * sizeof(double), hipMemcpyHostToDevice);	//Fill u using a_p elements

	//probably it will add overhead due to the multiple kernel calls
    	for(i=0; i<n; i++){
    	    	decomposition<<<dimGridLinear, dimBlockLinear>>>(d_l, d_u, n, i);
    	    	hipDeviceSynchronize();
    	}

	determinant <<<dimGridLinear, dimBlockLinear>>>(d_l, d_u, n, d_det, perm);

	hipMemcpy(&det, d_det, sizeof(double), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	printf("Determinant: %lf\n", det);
	if(det == 0.0){
		printf("ERROR: It is not possible to compute the inversion: the matrix is not squared\n");
		fclose(mat);
		hipFree(d_l);
		hipFree(d_u);
		hipFree(d_p);
		hipFree(d_a_inv);
		hipFree(d_det);
		hipFree(d_y);
		free(p);
		free(l);
		free(u);
		free(a_p);
		free(a_inv);
		free(m.mat);
		exit(1);
	}

	hipMemcpy(d_p, p, n * n * sizeof(double), hipMemcpyHostToDevice);

	inverse <<<dimGridLinear, dimBlockLinear>>>(d_l, d_p, d_u, d_a_inv, d_y, n);
	hipDeviceSynchronize();

	t = clock() - t;
	
	hipMemcpy(a_inv, d_a_inv, n*n*sizeof(double), hipMemcpyDeviceToHost);

	//hipProfilerStop();

	resultFile = fopen("inverse.txt", "w");
	printMatrix(a_inv, n, resultFile);

	printf("\nElapsed time: %lf seconds\n", ((double)t) / CLOCKS_PER_SEC);

	fclose(mat);
	fclose(resultFile);

	hipFree(d_l);
	hipFree(d_u);
	hipFree(d_p);
	hipFree(d_a_inv);
	hipFree(d_det);
	hipFree(d_y);
	free(p);
	free(l);
	free(u);
	free(a_p);
	free(a_inv);
	free(m.mat);

	return 0;
}
