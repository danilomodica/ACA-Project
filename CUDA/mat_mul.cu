#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include "hip/hip_runtime.h"
//#include "cuda_profiler_api.h"

#define THREADS 32 //In each block THREADS*THREADS threads

struct matrix {
	int ncols;
	int nrows;
	double* mat;
};

void readMatrix(struct matrix* m, FILE* file);
void printMatrix(struct matrix* m, FILE* file);
__global__ void matrixMul(double *d_m1, double *d_m2, double *d_m3, int row1, int row2, int col1, int col2);

/*
Knowing the number of rows and columns,
it reads a matrix from a file and stores it in the appropriate structure.
*/
void readMatrix(struct matrix* m, FILE* file) {
	int i, j;

	m->mat = (double*)malloc(m->ncols * m->nrows * sizeof(double));

	for (i = 0; i < m->nrows; i++) {
		for (j = 0; j < m->ncols; j++) {
			fscanf(file, "%lf", &m->mat[i * m->ncols + j]);
		}
	}
}

/*
The opposite operation to readMatrix. Saves a matrix in the file given as argument
*/
void printMatrix(struct matrix* m, FILE* file) {
	int i, j;

	for (i = 0; i < m->nrows; i++) {
		for (j = 0; j < m->ncols; j++) {
			fprintf(file, "%lf ", m->mat[i * m->ncols + j]);
		}
		fprintf(file, "\n");
	}
}

/*
Performs the multiplication operation between the matrices m1 and m2.
The result will be stored in the matrix m3.
*/
__global__ void matrixMul(double *d_m1, double *d_m2, double *d_m3, int row1, int row2, int col1, int col2){
	int i = blockIdx.y*blockDim.y+threadIdx.y;
	int j = blockIdx.x*blockDim.x+threadIdx.x;

	double sum = 0;
	int k;

	//the two previous for cycle are substituted by the matrix of threads
	if ((i < row1) && (j < col2)){
		for(k = 0; k<col1; k++){
			sum += d_m1[i*col1+k]*d_m2[k*col2+j];
		}
		d_m3[i*col2+j]=sum;
  }
}

int main(int argc, char* argv[]) {
	if(argc != 3){ //1- exe name, 2- mat1, 3- mat2
		printf("Parameter error.");
		exit(1);
	}

	FILE *mat1, *mat2, *resultFile;
	clock_t t;
	struct matrix m1, m2, m3;

	mat1 = fopen(argv[1], "r");
	mat2 = fopen(argv[2], "r");
	fscanf(mat1, "%d %d", &m1.nrows, &m1.ncols);
	fscanf(mat2, "%d %d", &m2.nrows, &m2.ncols);

	//Multiplication is permitted if m1 is m x n and m2 is n x p.
	if(m1.ncols != m2.nrows){
		printf("It is not possible to do matrix multiplication. Check matrices number of rows and cols.");
		fclose(mat1);
		fclose(mat2);
		exit(1);
	}

	readMatrix(&m1, mat1);
	readMatrix(&m2, mat2);

	//M3 initilization
	m3.nrows=m1.nrows;
	m3.ncols=m2.ncols;
	m3.mat = (double*)malloc(m3.ncols * m3.nrows * sizeof(double));

	//cudaProfilerStart();

	/* Device variable, allocations, and transfers */
	double *d_m1, *d_m2, *d_m3;
	hipMalloc((void**)&d_m1, m1.nrows*m1.ncols*sizeof(double));
	hipMalloc((void**)&d_m2, m2.nrows*m2.ncols*sizeof(double));
	hipMalloc((void**)&d_m3, m3.nrows*m3.ncols*sizeof(double));

	hipMemcpy(d_m1, m1.mat, m1.nrows*m1.ncols * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_m2, m2.mat, m2.nrows*m2.ncols * sizeof(double), hipMemcpyHostToDevice);

	hipMemset(d_m3, 0, m3.nrows*m3.ncols*sizeof(double));

	dim3 dimBlock(THREADS, THREADS);
	dim3 dimGrid((m2.ncols+dimBlock.x-1)/dimBlock.x, (m1.nrows+dimBlock.y-1)/dimBlock.y);

	t = clock();
	matrixMul <<<dimGrid, dimBlock>>>(d_m1, d_m2, d_m3, m1.nrows, m2.nrows, m1.ncols, m2.ncols);
	hipDeviceSynchronize();
	t = clock() - t; //total time spent in matrixMul

	hipMemcpy(m3.mat, d_m3, m3.nrows*m3.ncols * sizeof(double), hipMemcpyDeviceToHost);

	hipFree(d_m1);
	hipFree(d_m2);
	hipFree(d_m3);

	//cudaProfilerStop();

	resultFile = fopen("result.txt", "w");
	printMatrix(&m3, resultFile);

	printf("Elapsed time: %.5lf seconds", ((double)t)/CLOCKS_PER_SEC);

	fclose(mat1);
	fclose(mat2);
	fclose(resultFile);

	free(m1.mat);
	free(m2.mat);
	free(m3.mat);

	return 0;
}
