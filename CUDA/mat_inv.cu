#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include "hip/hip_runtime.h"
//#include "cuda_profiler_api.h"

#define THREADS 32

struct matrix {
	int ncols;
	int nrows;
	double* mat;
};

void readMatrix(struct matrix* m, FILE* file);
void printMatrix(double *squared_matrix, int n, FILE* file);
__global__ void determinant(double *l, double *u, int n, double *det, int perm);
__device__ void forwardSubstitution(double *d_l, double *d_p, double *d_y, int column, int n);
__device__ void backwardSubstitution(double *d_u, double *d_y, double *d_a_inv, int column, int n);
void pivoting(double *a, double *p, int n, int *perm);
void lu(double *l, double *u, int n);
__device__ double atomicMul(double* address, double val);
__global__ void fillInVectors(double *d_p, double *d_l, int n);
__global__ void inverse(double *d_l,double *d_p, double *d_u, double *d_a_inv, int n);
__host__ void checkCudaError(int linea);



/* Reads a matrix from a file and stores it into the appropriate structure. */
void readMatrix(struct matrix* m, FILE* file) {
	int i, j;

	m->mat = (double*)malloc(m->ncols * m->nrows * sizeof(double));

	for (i = 0; i < m->nrows; i++) {
		for (j = 0; j < m->ncols; j++) {
			fscanf(file, "%lf", &m->mat[i * m->ncols + j]);
		}
	}
}

/* Stores a matrix into the file passed as argument */
void printMatrix(double *squared_matrix, int n, FILE* file) {
	int i, j;

	for (i = 0; i < n; i++) {
		for (j = 0; j < n; j++) {
			fprintf(file, "%lf ", squared_matrix[i * n + j]);
		}
		fprintf(file, "\n");
	}
}

#if __CUDA_ARCH__ < 600
__device__ double atomicMul(double* address, double val){
    unsigned long long int* address_as_ull =
                              (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val *
                               __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}
#endif

__host__ void checkCudaError(int linea) {
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		printf("cuda error: %s alla linea %d\n", hipGetErrorString(err), linea);
		exit(-1);
	}
}

__global__ void fillInVectors(double *d_p, double *d_l, int n){
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	
	if(i<n){
		d_p[i * n + i]=1;
		d_l[i * n + i]=1;
	}
}

/* Becaute LU decomposition is used  det M = det LU = det L * det U, L and U are triangular
   so the determinant is calculated as the product of the diagonal elements
 */
__global__ void determinant(double *l, double *u, int n, double *d_det, int perm) {
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	double det = 1;

	if(i==0){
		d_det[0]=1;
	}
	__syncthreads();

	if(i<n){
		det *= l[i * n + i] * u[i * n + i];
	}

	atomicMul(d_det, det);
	d_det[0] = d_det[0]*pow(-1,perm);
}

/* Since L is a lower triangular matrix forward substitution is used to perform the calculus of Lx=y */
__device__ void forwardSubstitution(double *d_l, double *d_p, double *d_y, int column, int n) {
	int i, j;
	double sum = 0;

	for (i = 0; i < n; i++) {
	        for (j = 0; j < i; j++) {
	            sum = sum + d_l[i * n + j] * d_y[j];
	        }
	        d_y[i] = (d_p[i * n + column] - sum) / d_l[i * n + i];
	        sum = 0;
	}
}

__device__ void backwardSubstitution(double *d_u, double *d_y, double *d_a_inv, int column, int n) {
	int i, j;
	double sum;

	d_a_inv[(n-1)*n+column] = d_y[n-1] / d_u[(n-1) * n + (n-1)];
	
	for (i = n - 2; i >= 0; i--) {
		sum = d_y[i];
	        for (j = n - 1; j > i; j--) {
	           sum = sum - d_u[i * n + j] * d_a_inv[j*n+column];
	        }
	        d_a_inv[i*n+column] = sum / d_u[i * n + i];
	        sum = 0;
	}
}

/* Even if det(M)!=0, pivoting is performed to be sure that L and U are correctly upper and lower triangular matrix */
void pivoting(double *a, double *p, int n, int *perm) {
	int j, k;
	int isMaximum = 0;
	double *temp = (double*)malloc(n * sizeof(double));

	// k is column and j is row
	for (k = 0; k < n-1; k++) {
    		int imax = k;
        	for (j = k; j < n; j++) {
			if (a[j * n + k] > a[imax * n + k]) {  // finding the maximum index
				imax = j;
        		        isMaximum = 1;
        		}
        	}
        	if (isMaximum == 1) {
        		// swapping a[k] and a[imax]
			memcpy(temp, &a[k*n], n * sizeof(double));
			memcpy(&a[k*n], &a[imax*n], n * sizeof(double));
			memcpy(&a[imax*n], temp, n * sizeof(double));

			// swapping p[k] and p[imax]
			memcpy(temp, &p[k*n], n * sizeof(double));
			memcpy(&p[k*n], &p[imax*n], n * sizeof(double));
			memcpy(&p[imax*n], temp, n * sizeof(double));

			perm[0]++;
        		isMaximum = 0;
    		}
	}
	free(temp);
}

/* Perf LU decomposition of matrix M*/
void lu(double *l, double *u, int n) {
    int i, j, k;
    
	for (k = 0; k < n; k++) {
        	for (i = k + 1; i < n; i++) {
            			l[i * n + k] = u[i * n + k] / u[k * n + k];
            		for (j = k; j < n; j++) {
                		u[i * n + j] = u[i * n + j] - l[i * n + k] * u[k * n + j];
            		}
        	}
    	}
}

__global__ void inverse(double *d_l,double *d_p, double *d_u, double *d_a_inv, int n){
	int i = blockIdx.x*blockDim.x+threadIdx.x;	
	
	if(i<n){
		double *d_y = (double*)malloc(n*sizeof(double));
		memset(d_y, 0, n*sizeof(double));
	
		forwardSubstitution(d_l, d_p, d_y, i, n);
		backwardSubstitution(d_u, d_y, d_a_inv, i, n);
		
		free(d_y);
	}
}

int main(int argc, char* argv[]) {
	if(argc != 2) { //Checking parameters: 1.mat_inv.exe 2.matrix
		printf("Parameters error.\n");
		exit(1);
	}

	printf("This program compute the inverse of a squared matrix using only one thread\nPlease wait until computation are done...\n");

	FILE *mat, *resultFile;
	clock_t t;
	struct matrix m;
	int i, perm=0;

	mat = fopen(argv[1], "r");
	fscanf(mat, "%d %d", &m.nrows, &m.ncols);
	readMatrix(&m, mat);

	if (m.nrows != m.ncols) {
		printf("ERROR: It is not possible to compute the inversion: the matrix is not squared\n");
		fclose(mat);
		free(m.mat);
		exit(1);
	}

	int n = m.nrows; //matrix order (m is squared)

	//Create pivoting and inverse matrices
	double *a_inv = (double*)malloc(n * n * sizeof(double));
	double *p = (double*)malloc(n * n * sizeof(double));
	double *l = (double*)malloc(n * n * sizeof(double));
	double *a_p = (double*)malloc(n * n * sizeof(double));
	double *u = (double*)malloc(n * n * sizeof(double));
	double *y = (double*)malloc(n * sizeof(double));

	//Matrices initialization
	memset(a_inv, 0, n * n * sizeof(double));
	memcpy(a_p, m.mat, n * n * sizeof(double));


	/* Device variable, allocations, and transfers */
	double *d_l, *d_u, *d_p, *d_det, *d_a_inv;
	double det;

	hipMalloc((void**)&d_a_inv, n*n*sizeof(double));
	hipMalloc((void**)&d_p, n*n*sizeof(double));
	hipMalloc((void**)&d_l, n*n*sizeof(double));
	hipMalloc((void**)&d_u, n*n*sizeof(double));
   	hipMalloc((void**)&d_det, sizeof(double));

	hipMemset(d_p, 0, n * n * sizeof(double));
	hipMemset(d_l, 0, n * n * sizeof(double));

	int block_x = n / THREADS;
	if ((n) % THREADS != 0) {
		block_x++;
	}

	dim3 dimBlockLinear(THREADS, 1, 1);
	dim3 dimGridLinear(block_x, 1, 1);
	
	fillInVectors <<<dimGridLinear, dimBlockLinear>>>(d_p, d_l, n);
	
	hipMemcpy(l, d_l, n*n*sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(p, d_p, n*n*sizeof(double), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	
	t = clock();
	pivoting(a_p, p, n, &perm);

	memcpy(u, a_p, n * n * sizeof(double));	//Fill u using a_p elements

    	lu (l, u, n);
	
	hipMemcpy(d_l, l, n * n * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_u, u, n * n * sizeof(double), hipMemcpyHostToDevice);

	determinant <<<dimGridLinear, dimBlockLinear>>>(d_l, d_u, n, d_det, perm);

	hipMemcpy(&det, d_det, sizeof(double), hipMemcpyDeviceToHost);

	hipDeviceSynchronize();

	printf("Determinant: %lf\n", det);
	if(det == 0.0){
		printf("ERROR: It is not possible to compute the inversion: the matrix is not squared\n");
		fclose(mat);
		hipFree(d_l);
		hipFree(d_u);
		hipFree(d_p);
		hipFree(d_a_inv);
		hipFree(d_det);
		free(p);
		free(l);
		free(u);
		free(a_p);
		free(a_inv);
		free(m.mat);
		exit(1);
	}

	hipMemcpy(d_p, p, n * n * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_a_inv, a_inv, n * n * sizeof(double), hipMemcpyHostToDevice);

	inverse <<<dimGridLinear, dimBlockLinear>>>(d_l, d_p, d_u, d_a_inv, n);
	hipDeviceSynchronize();

	t = clock() - t;
	
	hipMemcpy(a_inv, d_a_inv, n*n*sizeof(double), hipMemcpyDeviceToHost);

	//cudaProfilerStop();

	resultFile = fopen("inverse.txt", "w");
	printMatrix(a_inv, n, resultFile);

	printf("\nElapsed time: %lf seconds\n", ((double)t) / CLOCKS_PER_SEC);

	fclose(mat);
	fclose(resultFile);

	hipFree(d_l);
	hipFree(d_u);
	hipFree(d_p);
	hipFree(d_a_inv);
	hipFree(d_det);

	free(y);
	free(p);
	free(l);
	free(u);
	free(a_p);
	free(a_inv);
	free(m.mat);

	return 0;
}
